
#include <hip/hip_runtime.h>
#define EMPTY 0
#define RED 1
#define BLUE 2

__global__ void init_kernel(int * domain, int domain_x)
{
	// Dummy initialization
	domain[blockIdx.y * domain_x + blockIdx.x * blockDim.x + threadIdx.x]
		= ((blockIdx.x+threadIdx.x) == 0 ? 1 : 0);
		//= (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3;
}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy, unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx) % domain_x;	// Wrap around
    y = (unsigned int)(y + dy) % domain_y;
    return source_domain[y * domain_x + x];
}

__device__ void inc_color(int cell, int *nb_blue, int *nb_red) {
	if (cell == BLUE) (*nb_blue)++;
	else if (cell == RED) (*nb_red)++;
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain, int domain_x, int domain_y)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
	int pos_x = threadIdx.x + 1, pos_y = threadIdx.y + 1;
	__shared__ int shared_bloc[18][10];

	// Read cell
    shared_bloc[pos_x][pos_y] = read_cell(source_domain, tx, ty, 0, 0, domain_x, domain_y);

	if (pos_x == 1) 
		shared_bloc[pos_x-1][pos_y] = read_cell(source_domain, tx, ty, -1, 0, domain_x, domain_y);
	else if (pos_x == 16) 
		shared_bloc[pos_x+1][pos_y] = read_cell(source_domain, tx, ty, 1, 0, domain_x, domain_y);

	if (pos_y == 1) 
		shared_bloc[pos_x][pos_y-1] = read_cell(source_domain, tx, ty, 0, -1, domain_x, domain_y);
	else if (pos_y == 8) 
		shared_bloc[pos_x][pos_y+1] = read_cell(source_domain, tx, ty, 0, 1, domain_x, domain_y);

	if (pos_x == 1 && pos_y == 1)
		shared_bloc[pos_x-1][pos_y-1] = read_cell(source_domain, tx, ty, -1, -1, domain_x, domain_y);
	else if (pos_x == 1 && pos_y == 8)
		shared_bloc[pos_x-1][pos_y+1] = read_cell(source_domain, tx, ty, -1, 1, domain_x, domain_y);
	else if (pos_x == 16 && pos_y == 1)
		shared_bloc[pos_x+1][pos_y-1] = read_cell(source_domain, tx, ty, 1, -1, domain_x, domain_y);
	else if (pos_x == 16 && pos_y == 8)
		shared_bloc[pos_x+1][pos_y+1] = read_cell(source_domain, tx, ty, 1, 1, domain_x, domain_y);
    
    // Read the 8 neighbors and count number of blue and red
	int nb_blue = 0, nb_red = 0;

	__syncthreads();

	inc_color(shared_bloc[pos_x][pos_y-1], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x][pos_y+1], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x-1][pos_y], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x+1][pos_y], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x+1][pos_y-1], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x-1][pos_y+1], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x-1][pos_y-1], &nb_blue, &nb_red);
	inc_color(shared_bloc[pos_x+1][pos_y+1], &nb_blue, &nb_red);

	// Compute new value
	int res = 0;

	if (shared_bloc[pos_x][pos_y] != 0 && (nb_blue + nb_red == 2 || nb_blue + nb_red == 3))
		res = shared_bloc[pos_x][pos_y];
	else if (shared_bloc[pos_x][pos_y] == 0 && nb_blue + nb_red == 3) {
		if (nb_blue > nb_red)
			res = BLUE;
		else
			res = RED;
	}
	
	// Write it in dest_domain
	dest_domain[ty * domain_x + tx] = res;
}